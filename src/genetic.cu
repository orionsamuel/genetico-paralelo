#include "hip/hip_runtime.h"
#include "../include/genetic.hpp"
#include "hip/hip_runtime.h"

__global__ void FillPopulation(individual *d_population, int size_population, int n_permeability){
    int i = threadIdx.x;
    int j = threadIdx.y;
    
    if((i < size_population) && (j < n_permeability)){
        d_population[i].permeability_x[j] = 0;
        d_population[i].permeability_y[j] = 0;
        d_population[i].permeability_z[j] = 0;
        d_population[i].porosity = 0;
        d_population[i].error_rank = 0;
    }
}

__global__ void FillChildren(individual* d_children, int crossover_rate, int n_permeability){
    int i = threadIdx.x;
    int j = threadIdx.y;

    if((i < crossover_rate) && (j < n_permeability)){
        d_children[i].porosity = 0;
        d_children[i].error_rank = 0;
        d_children[i].permeability_x[j] = 0;
        d_children[i].permeability_y[j] = 0;
        d_children[i].permeability_z[j] = 0;
    }
}

genetic_algorithm::genetic_algorithm(){
    srand (time(0));
}

genetic_algorithm::~genetic_algorithm(){

}

void genetic_algorithm::FirstPopulation(){
    srand((unsigned)time(0));
    
    CreateResultDir(0);

    individual *d_population;

    this->population.resize(SIZE_POPULATION);

    dim3 blocks(SIZE_POPULATION,N_PERMEABILITY);

    hipMalloc((void **)&d_population, SIZE_POPULATION * sizeof(individual));

    hipMemcpy(d_population, this->population.data(), SIZE_POPULATION * sizeof(individual), hipMemcpyHostToDevice);

    FillPopulation<<<1,blocks>>>(d_population, SIZE_POPULATION, N_PERMEABILITY);
    hipDeviceSynchronize();

    hipMemcpy(this->population.data(), &d_population, SIZE_POPULATION * sizeof(individual), hipMemcpyDeviceToHost);

    hipFree(d_population);

    for(int i = 0; i < SIZE_POPULATION; i++){
        this->population[i].porosity = Rand_double(MIN_POROSITY, MAX_POROSITY);
        #pragma omp parallel for
        for(int j = 0; j < N_PERMEABILITY; j++){
            this->population[i].permeability_x[j] = Rand_double(MIN_PERMEABILITY, MAX_PERMEABILITY);
            this->population[i].permeability_y[j] = Rand_double(MIN_PERMEABILITY, MAX_PERMEABILITY);
            this->population[i].permeability_z[j] = Rand_double(MIN_PERMEABILITY, MAX_PERMEABILITY);
        }
    }
 
    #pragma omp parallel for
    for(int i = 0; i < SIZE_POPULATION; i++){
        WriteSimulationFile(0, i, simulationFile, fileName, population);
    }
    
    Simulation(0, fileName);
    Fitness(0);
    sort(begin(this->population), end(this->population), Compare);
  
    WriteErrorFile(0, population);

    #pragma omp parallel for
    for(int i = 0; i < SIZE_POPULATION; i++){
        WriteSimulationFile(0, i, simulationFile, fileName, population);
    }
      
}

void genetic_algorithm::OtherPopulations(int idIteration){
    Crossover();

    CreateResultDir(idIteration);

    #pragma omp parallel for
    for(int i = 0; i < SIZE_POPULATION; i++){
        system(Command("cp ../Output/"+to_string(idIteration-1)+"/"+to_string(i)+"-"+fileName+".DATA ../Output/"+to_string(idIteration)+"/"+to_string(i)+"-"+fileName+".DATA"));
    }

    #pragma omp parallel for
    for(int i = SIZE_POPULATION; i < (SIZE_POPULATION + this->crossover_rate); i++){
        WriteSimulationFile(idIteration, i, simulationFile, fileName, population);
    }

    Simulation(idIteration, fileName);
    Fitness(idIteration);
    sort(begin(this->population), end(this->population), Compare);

    WriteErrorFile(idIteration, population);

    system(Command("rm -f ../Output/"+to_string(idIteration)+"/*.DATA"));

    #pragma omp parallel for
    for(int i = SIZE_POPULATION; i < (SIZE_POPULATION + this->crossover_rate); i++){
        this->population.pop_back();
    }

    #pragma omp parallel for
    for(int i = 0; i < SIZE_POPULATION; i++){
        WriteSimulationFile(idIteration, i, simulationFile, fileName, population);
    }

}

void genetic_algorithm::Fitness(int idIteration){
    if(idIteration == 0){
        #pragma omp parallel for
        for(int i = 0; i < SIZE_POPULATION; i++){
            string oilOutputResult = "../Output/"+to_string(idIteration)+"/oleo/"+to_string(i)+".txt";
            string waterOutputResult = "../Output/"+to_string(idIteration)+"/agua/"+to_string(i)+".txt";
            string gasOutputResult = "../Output/"+to_string(idIteration)+"/gas/"+to_string(i)+".txt";
            this->population[i].error_rank = activationFunction(waterOutputResult, oilOutputResult, gasOutputResult, realResults, idIteration, i);
        }
    }else{
        #pragma omp parallel for
        for(int i = SIZE_POPULATION; i < (SIZE_POPULATION + this->crossover_rate); i++){
            string oilOutputResult = "../Output/"+to_string(idIteration)+"/oleo/"+to_string(i)+".txt";
            string waterOutputResult = "../Output/"+to_string(idIteration)+"/agua/"+to_string(i)+".txt";
            string gasOutputResult = "../Output/"+to_string(idIteration)+"/gas/"+to_string(i)+".txt";
            this->population[i].error_rank = activationFunction(waterOutputResult, oilOutputResult, gasOutputResult, realResults, idIteration, i);
        }
    }
}

void genetic_algorithm::Crossover(){
    individual *d_children;

    dim3 blocks(crossover_rate,N_PERMEABILITY);

    hipMalloc((void **)&d_children, crossover_rate * sizeof(individual));

    hipMemcpy(d_children, &this->children, crossover_rate * sizeof(individual), hipMemcpyHostToDevice);

    FillChildren<<<1,blocks>>>(d_children, crossover_rate, N_PERMEABILITY);
    hipDeviceSynchronize();

    hipMemcpy(&this->children, &d_children, crossover_rate * sizeof(individual), hipMemcpyDeviceToHost);

    hipFree(d_children);

    for(int i = 0; i < crossover_rate; i++){
        this->children[i].porosity = 0;
        this->children[i].error_rank = 0;
        #pragma omp parallel for
        for(int j = 0; j < N_PERMEABILITY; j++){
            this->children[i].permeability_x[j] = 0;
            this->children[i].permeability_y[j] = 0;
            this->children[i].permeability_z[j] = 0;
        }
    }

    int percent = rand() % 2;
    if(percent == 0){
        int count = 0;
        while(count < crossover_rate){
            this->children[count].porosity = this->population[count].porosity;
            this->children[count + 1].porosity = this->population[count + 1].porosity;
            #pragma omp parallel for
            for(int i = 0; i < N_PERMEABILITY; i++){
                this->children[count].permeability_x[i] = this->population[count + 1].permeability_x[i];
                this->children[count].permeability_y[i] = this->population[count + 1].permeability_y[i];
                this->children[count].permeability_z[i] = this->population[count + 1].permeability_z[i];

                this->children[count + 1].permeability_x[i] = this->population[count].permeability_x[i];
                this->children[count + 1].permeability_y[i] = this->population[count].permeability_y[i];
                this->children[count + 1].permeability_z[i] = this->population[count].permeability_z[i];
            }
            count = count + 2;
        }
    }else {
        int count = 0;
        while(count < crossover_rate){
            this->children[count].porosity = this->population[count].porosity;
            this->children[count + 1].porosity = this->population[count + 1].porosity;
            #pragma omp parallel for
            for(int i = 0; i < N_PERMEABILITY; i++){
                this->children[count].permeability_x[i] = this->population[count].permeability_x[i];
                this->children[count].permeability_y[i] = this->population[count + 1].permeability_y[i];
                this->children[count].permeability_z[i] = this->population[count + 1].permeability_z[i];

                this->children[count + 1].permeability_x[i] = this->population[count + 1].permeability_x[i];
                this->children[count + 1].permeability_y[i] = this->population[count].permeability_y[i];
                this->children[count + 1].permeability_z[i] = this->population[count].permeability_z[i];
            }
            count = count + 2;
        }
    }

    //Mutation();

    for(int i = 0; i < crossover_rate; i++){
        this->children[i].porosity = floor(this->children[i].porosity * 100) / 100;
        #pragma omp parallel for
        for(int j = 0; j < N_PERMEABILITY; j++){
            this->children[i].permeability_x[j] = floor(this->children[i].permeability_x[j] * 100) / 100;
            this->children[i].permeability_y[j] = floor(this->children[i].permeability_y[j] * 100) / 100;
            this->children[i].permeability_z[j] = floor(this->children[i].permeability_z[j] * 100) / 100;
        }
    }
    
    for(int i = 0; i < this->crossover_rate; i++){
        this->population.push_back(children[i]);
    }
}

void genetic_algorithm::Mutation(){
    for(int i = 0; i < this->mutation_rate; i++){
        int gene = rand() % 3;
        int tunning = rand() % 2;

        mutationValue newValue;

        if(tunning == 0){
            newValue = RandMutationValue(this->children[i], gene, true);
        }else{
            newValue = RandMutationValue(this->children[i], gene, false);
        }

        this->children[i].porosity = newValue.porosity;
        this->children[i].permeability_x[gene] = newValue.permeability_x;
        this->children[i].permeability_y[gene] = newValue.permeability_y;
        this->children[i].permeability_z[gene] = newValue.permeability_z;
        
    }
}

void genetic_algorithm::Init(){
    CreateOutputDir();
    
    string oilInputResult = ReadFileInput(inputOil);
    string waterInputResult = ReadFileInput(inputWater);
    string gasInputResult = ReadFileInput(inputGas);
    
    realResults = ConvertStringInputToDoubleResult(waterInputResult, oilInputResult, gasInputResult);    

    FirstPopulation();
    int count = 1;
    while(count < N_GENERATIONS){
        OtherPopulations(count);
        count++;
    }

}